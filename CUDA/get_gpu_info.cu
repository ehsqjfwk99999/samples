#include <hip/hip_runtime.h>
#include <stdio.h>

#define RUNTIME_API_CALL(apiFuncCall)                                          \
  do {                                                                         \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",     \
              __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));  \
      exit(-1);                                                                \
    }                                                                          \
  } while (0)

int main() {
  int device_count;
  int current_device;
  hipDeviceProp_t devProp;

  RUNTIME_API_CALL(hipGetDeviceCount(&device_count));
  RUNTIME_API_CALL(hipGetDevice(&current_device));

  printf("\nNumber of devices: %d\n", device_count);
  for (int i = 0; i < device_count; i++) {
    RUNTIME_API_CALL(hipGetDeviceProperties(&devProp, i));
    if (i == current_device) {
      printf("\nDevice: %d (Current Device)\n", i);
    } else {
      printf("\nDevice: %d\n", i);
    }
    printf("- Device Name: %s\n", devProp.name);
    printf("- Compute Capability: %d.%d\n", devProp.major, devProp.minor);
    printf("- Max Grid Size: (%d, %d, %d)\n", devProp.maxGridSize[0],
           devProp.maxGridSize[1], devProp.maxGridSize[2]);
    printf("- Max Block Size: (%d, %d, %d)\n", devProp.maxThreadsDim[0],
           devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
    printf("- Max Threads per Block: %d\n", devProp.maxThreadsPerBlock);
    printf("- Warp Size: %d\n", devProp.warpSize);
  }

  printf("\n");
  return 0;
}
