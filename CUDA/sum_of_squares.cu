#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 2048
#define THREADS_PER_BLOCK 1024

__global__ void kernel(int *a, int *b, unsigned *c) {
  __shared__ int temp[THREADS_PER_BLOCK];

  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  temp[threadIdx.x] = a[idx] * b[idx];

  __syncthreads();

  if (threadIdx.x == 0) {
    unsigned sum = 0;
    for (int i = 0; i < THREADS_PER_BLOCK; i++) {
      sum += temp[i];
    }
    atomicAdd(c, sum);
  }
}

int main() {
  int *a, *b;
  int *d_a, *d_b;
  unsigned *d_c;
  int malloc_size = sizeof(int) * SIZE;
  unsigned total_sum;

  a = (int *)malloc(malloc_size);
  b = (int *)malloc(malloc_size);

  hipMalloc(&d_a, malloc_size);
  hipMalloc(&d_b, malloc_size);
  hipMalloc(&d_c, sizeof(unsigned));

  for (int i = 0; i < SIZE; i++) {
    a[i] = b[i] = i + 1;
  }

  hipMemcpy(d_a, a, malloc_size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, malloc_size, hipMemcpyHostToDevice);

  kernel<<<SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

  hipMemcpy(&total_sum, d_c, sizeof(unsigned), hipMemcpyDeviceToHost);

  printf("Total Sum: %u\n", total_sum);

  free(a);
  free(b);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}